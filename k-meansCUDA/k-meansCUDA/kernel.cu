#include "hip/hip_runtime.h"
﻿//#include "baseKernel.h"
//#include "simpleKernels.cu"
//#include "atomicKernels.cu"
//#include "manyDimensionsKernels.cu";

#include "hip/hip_runtime.h"
#include ""

#include <stdint.h>
#include <stdio.h> 
#include <sstream>
#include <iostream>

#include <time.h>

#include <stdlib.h>
#include <vector>

uint64_t dimension;
typedef float value_t;
typedef unsigned char cluster_t;

hipError_t countKMeansSimple(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize, value_t* means, uint32_t* assignedClusters, uint64_t dimension);
hipError_t countKMeansAtomic(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize, value_t* means, uint32_t* assignedClusters, uint64_t dimension);
hipError_t countKMeansManyMeans(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize, value_t* means, uint32_t* assignedClusters, uint64_t dimension);

void usage()
{
	std::cout << "Usage:" << std::endl << "kmeans <data_file> <means_file> <clusters_file> <k> <iterations>" << std::endl << "kmeans --generate <data_file> <size> <seed>" << std::endl;
}

#pragma region Kernels
__global__ void findNearestClusterKernel(const uint32_t meansSize, const value_t *means, const uint32_t dataSize, const value_t* data, uint32_t* assignedClusters, const uint32_t dimension)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    value_t minDistance = LLONG_MAX, distance = 0, difference = 0;
    for (size_t i = 0; i < meansSize; ++i)
    {
        distance = 0;
        for (size_t j = 0; j < dimension; ++j)
        {
            difference = means[i * dimension + j] - data[id * dimension + j];
            distance += difference * difference;
        }
        if (minDistance > distance)
        {
            minDistance = distance;
            assignedClusters[id] = i;
        }
    }
}

__global__ void countNewMeansKernel(uint32_t* assignedClusters, const uint32_t dataSize, const value_t* data, value_t* means, const uint32_t dimension, uint32_t* test)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int idOffset = id * dimension;
    uint32_t count = 0;
    for (size_t i = idOffset; i < idOffset + dimension; ++i)
    {
        means[i] = 0;
    }
    for (size_t i = 0; i < dataSize; ++i)
    {
        if (assignedClusters[i] == id)
        {
            for (size_t j = 0; j < dimension; ++j)
            {
                means[idOffset + j] += data[i * dimension + j];
            }
            ++count;
        }
    }
    for (size_t i = idOffset; i < idOffset + dimension; ++i)
    {
        means[i] /= count;
    }
    test[id] = count;
}

__global__ void findNearestClusterAtomicKernel(const uint32_t meansSize, const value_t *means, value_t *measnSums, const uint32_t dataSize, const value_t* data, uint32_t* counts, uint32_t* assignedClusters, const uint32_t dimension)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    value_t minDistance = LLONG_MAX, distance = 0, difference = 0;
    int clusterID = -1;
    for (size_t i = 0; i < meansSize; ++i)
    {
        distance = 0;
        for (size_t j = 0; j < dimension; ++j)
        {
            difference = means[i * dimension + j] - data[id * dimension + j];
            distance += difference * difference;
        }
        if (minDistance > distance)
        {
            minDistance = distance;
            clusterID = i;
        }
    }
    atomicInc(&counts[clusterID], INT32_MAX);
    assignedClusters[id] = clusterID;
    for (size_t j = 0; j < dimension; ++j)
    {
        atomicAdd(&measnSums[clusterID * dimension + j], data[id * dimension + j]);
    }
}

__global__ void countDivMeansKernel(const uint32_t meansSize, const uint32_t* counts, value_t* means, const value_t* meansSums, const uint32_t dimension)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    means[id] = meansSums[id] / (value_t)counts[blockIdx.x];
}

__global__ void findNearestClusterManyDimKernel(const uint32_t meansSize, const value_t *means, value_t *measnSums, const uint32_t dataSize, const value_t* data, uint32_t* counts, uint32_t* assignedClusters, const uint32_t dimension)
{
    //int id = threadIdx.x;
    value_t minDistance = LLONG_MAX, difference = 0;
    int clusterID = -1;
    extern __shared__ value_t distances[];

    for (size_t i = 0; i < meansSize; ++i)
    {
        difference = means[i * dimension + threadIdx.x] - data[blockIdx.x * dimension + threadIdx.x];
        distances[threadIdx.x] = difference * difference;
        //sum distances in block
        __syncthreads();
        for (size_t j = dimension / 2; j > 0; j >>= 1)
        {
            if (threadIdx.x < j)
            {
                distances[threadIdx.x] += distances[threadIdx.x + j];
            }
            __syncthreads();
        }

        if ( (minDistance > distances[0]))
        {
            minDistance = distances[0];
            clusterID = i;
        }
    }

    if (threadIdx.x == 0)
    {
        atomicInc(&counts[clusterID], INT32_MAX);
        assignedClusters[blockIdx.x] = clusterID;
    }

    atomicAdd(&measnSums[clusterID * dimension + threadIdx.x], data[blockIdx.x * dimension + threadIdx.x]);
}
#pragma endregion

value_t* load(const std::string& file_name, uint64_t& dataSize)
{
	FILE* f = fopen(file_name.c_str(), "rb");
	if (!f) throw std::runtime_error("cannot open file for reading");
	//if (fseek(f, 0, SEEK_END)) throw std::runtime_error("seeking failed");
	if (!fread(&dataSize, sizeof(uint64_t), 1, f))  throw std::runtime_error("size cannot be read");
	if (!fread(&dimension, sizeof(uint64_t), 1, f))  throw std::runtime_error("dimension cannot be read");
	value_t* data = (value_t*)calloc(dataSize * dimension, sizeof(value_t));
	if (!fread(data, sizeof(value_t), dataSize * dimension, f))  throw std::runtime_error("value cannot be read");
	return data;
}

template<typename T>
T lexical_cast(const std::string& x)
{
	std::istringstream stream(x);
	T res;
	stream >> res;
	return res;
}

void save_results(const std::string& means_file_name, const std::string& clusters_file_name, const uint32_t meansSize, const value_t* means, const uint32_t dataSize, const value_t* data, const uint32_t* assignedClusters)
{
	FILE* f = fopen(means_file_name.c_str(), "wb");
	if (!f) throw std::runtime_error("cannot open file for writing");
	if (!fwrite(&dimension, sizeof(uint64_t), 1, f)) throw std::runtime_error("dimension cannot be written");
	//if (!fwrite(means, sizeof(value_t), dimension * meansSize, f)) throw std::runtime_error("value cannot be written");
	for (size_t i = 0; i < meansSize; i++)
	{
		if (!fwrite(&means[i*dimension], sizeof(value_t), dimension, f)) throw std::runtime_error("value cannot be written");
		if (!fwrite(&i, sizeof(unsigned char), 1, f)) throw std::runtime_error("value cannot be written");
	}
	if (fclose(f)) throw std::runtime_error("closing the file failed");


	f = fopen(clusters_file_name.c_str(), "wb");
	if (!f) throw std::runtime_error("cannot open file for writing");
	if (!fwrite(&dimension, sizeof(uint64_t), 1, f)) throw std::runtime_error("dimension cannot be written");
	for (size_t i = 0; i < dataSize; i++)
	{
		if (!fwrite(&data[i*dimension], sizeof(value_t), dimension, f)) throw std::runtime_error("value cannot be written");
		if (!fwrite(&assignedClusters[i], sizeof(unsigned char), 1, f)) throw std::runtime_error("value cannot be written");
		//if (!fwrite(&i, sizeof(value_t), 1, f)) throw std::runtime_error("distance cannot be written");
	}
	if (fclose(f)) throw std::runtime_error("closing the file failed");
}

int main(int argc, const char* argv[])
{
	if (argc == 6)
	{
		std::string file_name(argv[1]);
		std::string means_file_name(argv[2]);
		std::string clusters_file_name(argv[3]);
		std::string s_k(argv[4]);
		std::string s_iterations(argv[5]);
		uint32_t k = lexical_cast<uint32_t>(s_k);
		uint32_t iterations = lexical_cast<uint32_t>(s_iterations);
		uint64_t dataSize;

		value_t* data = load(file_name, dataSize);
		value_t* means = (value_t*)calloc(k * dimension, sizeof(value_t));
		uint32_t* assignedClusters = (uint32_t*)calloc(dataSize * dimension, sizeof(uint32_t));
		memcpy(means, data, k * dimension * sizeof(value_t));

		// Add vectors in parallel.
        hipError_t cudaStatus = countKMeansManyMeans(iterations, dataSize, data, k, means, assignedClusters, dimension);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		save_results(means_file_name, clusters_file_name, k, means, dataSize, data, assignedClusters);

		free(data);
		free(means);
		free(assignedClusters);

		return 0;
	}
	usage();
	return 1;
}

#pragma region Tasks
hipError_t countKMeansSimple(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize, value_t* means, uint32_t* assignedClusters, uint64_t dimension)
{
    value_t* dev_means = 0;
    value_t* dev_data = 0;
    uint32_t* dev_assignedClusters = 0, *dev_test = 0;
    hipError_t cudaStatus;

    clock_t start, end;
    start = clock();

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_means, meansSize * dimension * sizeof(value_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_data, dataSize * dimension * sizeof(value_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_assignedClusters, dataSize * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_test, meansSize * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_means, means, meansSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_data, data, dataSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    uint32_t* test = (uint32_t*)calloc(meansSize, sizeof(uint32_t));

    // Launch a kernel on the GPU with one thread for each element.
    int blockSizeN = 32;
    int nBlocksN = (dataSize - 1) / blockSizeN + 1;
    int blockSizeM = 16;
    int nBlocksM = (meansSize - 1) / blockSizeM + 1;
    for (uint32_t i = 0; i < iterations; ++i)
    {
        findNearestClusterKernel << <nBlocksN, blockSizeN >> >(meansSize, dev_means, dataSize, dev_data, dev_assignedClusters, dimension);
        hipDeviceSynchronize();
        countNewMeansKernel << <nBlocksM, blockSizeM >> >(dev_assignedClusters, dataSize, dev_data, dev_means, dimension, dev_test);
        hipDeviceSynchronize();
        //cudaStatus = hipMemcpy(test, dev_test, meansSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
        //std::vector<uint32_t> t(test, test + meansSize);
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(means, dev_means, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(assignedClusters, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



Error:
    hipFree(dev_data);
    hipFree(dev_means);
    hipFree(dev_assignedClusters);

    end = clock();
    std::cout << "Time required for execution: "
        << (double)(end - start) / CLOCKS_PER_SEC
        << " seconds." << "\n\n";

    return cudaStatus;
}

hipError_t countKMeansAtomic(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize, value_t* means, uint32_t* assignedClusters, uint64_t dimension)
{
    value_t* dev_means = 0, *dev_data = 0, *dev_meansSums = 0, *dev_temp = 0;
    uint32_t* dev_assignedClusters = 0, *dev_counts = 0;
    hipError_t cudaStatus;

    clock_t start, end;
    start = clock();

    //std::vector<uint32_t> testVector(meansSize);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_means, meansSize * dimension * sizeof(value_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_meansSums, meansSize * dimension * sizeof(value_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    else
    {
        hipMemset(dev_meansSums, 0, meansSize * dimension * sizeof(value_t));
    }

    cudaStatus = hipMalloc((void**)&dev_data, dataSize * dimension * sizeof(value_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_assignedClusters, dataSize * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_counts, meansSize * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    else
    {
        hipMemset(dev_counts, 0, meansSize * sizeof(uint32_t));
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_means, means, meansSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_data, data, dataSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //uint32_t* test = (uint32_t*)calloc(meansSize, sizeof(uint32_t));
    //value_t* testMeans = (value_t*)calloc(meansSize * dimension , sizeof(value_t));

    // Launch a kernel on the GPU with one thread for each element.
    int blockSizeN = 32;
    int nBlocksN = (dataSize - 1) / blockSizeN + 1;
    //int blockSizeM = 16;
    //int nBlocksM = (meansSize - 1) / blockSizeM + 1;
    for (uint32_t i = 0; i < iterations; ++i)
    {
        findNearestClusterAtomicKernel << <nBlocksN, blockSizeN >> >(meansSize, dev_means, dev_meansSums, dataSize, dev_data, dev_counts, dev_assignedClusters, dimension);
        hipDeviceSynchronize();
        countDivMeansKernel << <meansSize, dimension >> >(meansSize, dev_counts, dev_means, dev_meansSums, dimension);
        hipDeviceSynchronize();

        hipMemset(dev_meansSums, 0, meansSize * dimension * sizeof(value_t));
        hipMemset(dev_counts, 0, meansSize * sizeof(uint32_t));
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(means, dev_means, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(assignedClusters, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



Error:
    hipFree(dev_data);
    hipFree(dev_means);
    hipFree(dev_meansSums);
    hipFree(dev_assignedClusters);
    hipFree(dev_counts);


    end = clock();
    std::cout << "Time required for execution: "
        << (double)(end - start) / CLOCKS_PER_SEC
        << " seconds." << "\n\n";

    return cudaStatus;
}

hipError_t countKMeansManyMeans(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize, value_t* means, uint32_t* assignedClusters, uint64_t dimension)
{
    value_t* dev_means = 0, *dev_data = 0, *dev_meansSums = 0, *dev_temp = 0;
    uint32_t* dev_assignedClusters = 0, *dev_counts = 0;
    hipError_t cudaStatus;

    //uint32_t* testAssigned,* testCounts;
    //value_t* testDistances;

    //testAssigned = (uint32_t*)malloc(dataSize * sizeof(uint32_t));
    //testCounts = (uint32_t*)malloc(meansSize * sizeof(uint32_t));
    //testDistances = (value_t*)malloc(meansSize * dimension * sizeof(value_t));

    clock_t start, end;
    start = clock();

    //std::vector<uint32_t> testVector(meansSize);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_means, meansSize * dimension * sizeof(value_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_meansSums, meansSize * dimension * sizeof(value_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    else
    {
        hipMemset(dev_meansSums, 0, meansSize * dimension * sizeof(value_t));
    }

    cudaStatus = hipMalloc((void**)&dev_data, dataSize * dimension * sizeof(value_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_assignedClusters, dataSize * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_counts, meansSize * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    else
    {
        hipMemset(dev_counts, 0, meansSize * sizeof(uint32_t));
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_means, means, meansSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_data, data, dataSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //uint32_t* test = (uint32_t*)calloc(meansSize, sizeof(uint32_t));
    //value_t* testMeans = (value_t*)calloc(meansSize * dimension , sizeof(value_t));

    // Launch a kernel on the GPU with one thread for each element.
    int blockSizeN = 32;
    int nBlocksN = (dataSize - 1) / blockSizeN + 1;
    //int blockSizeM = 16;
    //int nBlocksM = (meansSize - 1) / blockSizeM + 1;
    for (uint32_t i = 0; i < iterations; ++i)
    {
        findNearestClusterManyDimKernel << <dataSize, dimension, sizeof(value_t) * dimension >> >(meansSize, dev_means, dev_meansSums, dataSize, dev_data, dev_counts, dev_assignedClusters, dimension);
        hipDeviceSynchronize();
        //hipMemcpy(testAssigned, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
        //std::vector<uint32_t> t(testAssigned, testAssigned + dataSize);
        //hipMemcpy(testDistances, dev_meansSums, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
        //std::vector<uint32_t> t2(testDistances, testDistances + meansSize);
        //hipMemcpy(testCounts, dev_counts, meansSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
        //std::vector<uint32_t> t3(testCounts, testCounts+ meansSize);
        countDivMeansKernel << <meansSize, dimension >> >(meansSize, dev_counts, dev_means, dev_meansSums, dimension);
        hipDeviceSynchronize();

        hipMemset(dev_meansSums, 0, meansSize * dimension * sizeof(value_t));
        hipMemset(dev_counts, 0, meansSize * sizeof(uint32_t));
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(means, dev_means, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(assignedClusters, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



Error:
    hipFree(dev_data);
    hipFree(dev_means);
    hipFree(dev_meansSums);
    hipFree(dev_assignedClusters);
    hipFree(dev_counts);


    end = clock();
    std::cout << "Time required for execution: "
        << (double)(end - start) / CLOCKS_PER_SEC
        << " seconds." << "\n\n";

    return cudaStatus;
}

#pragma endregion Tasks