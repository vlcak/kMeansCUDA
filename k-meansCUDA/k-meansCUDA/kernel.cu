#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <sstream>
#include <iostream>

#include <vector>

uint64_t dimension;
typedef float value_t;
typedef unsigned char cluster_t;

hipError_t countKMeans(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize, value_t* means, uint32_t* assignedClusters);

__global__ void findNearestClusterKernel(const uint32_t meansSize, const value_t *means, const uint32_t dataSize, const value_t* data, uint32_t* assignedClusters, const uint32_t dimension)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
	value_t minDistance = LLONG_MAX, distance = 0, difference = 0;
	for (size_t i = 0; i < meansSize; ++i)
	{
		distance = 0;
		for (size_t j = 0; j < dimension; ++j)
		{
			difference = means[i * dimension + j] - data[id * dimension + j];
			distance += difference * difference;
		}
		if (minDistance > distance)
		{
			minDistance = distance;
			assignedClusters[id] = i;
		}
	}
}

__global__ void countNewMeansKernel(uint32_t* assignedClusters, const uint32_t dataSize, const value_t* data, value_t* means, const uint32_t dimension, uint32_t* test)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int idOffset = id * dimension;
	uint32_t count = 0;
	for (size_t i = idOffset; i < idOffset + dimension; ++i)
	{
		means[i] = 0;
	}
	for (size_t i = 0; i < dataSize; ++i)
	{
		if (assignedClusters[i] == id)
		{
			for (size_t j = 0; j < dimension; ++j)
			{
				means[idOffset + j] += data[i * dimension + j];
			}
			++count;
		}
	}
	for (size_t i = idOffset; i < idOffset + dimension; ++i)
	{
		means[i] /= count;
	}
	test[id] = count;
}

void usage()
{
	std::cout << "Usage:" << std::endl << "kmeans <data_file> <means_file> <clusters_file> <k> <iterations>" << std::endl << "kmeans --generate <data_file> <size> <seed>" << std::endl;
}

value_t* load(const std::string& file_name, uint64_t& dataSize)
{
	FILE* f = fopen(file_name.c_str(), "rb");
	if (!f) throw std::runtime_error("cannot open file for reading");
	//if (fseek(f, 0, SEEK_END)) throw std::runtime_error("seeking failed");
	if (!fread(&dataSize, sizeof(uint64_t), 1, f))  throw std::runtime_error("size cannot be read");
	if (!fread(&dimension, sizeof(uint64_t), 1, f))  throw std::runtime_error("dimension cannot be read");
	value_t* data = (value_t*)calloc(dataSize * dimension, sizeof(value_t));
	if (!fread(data, sizeof(value_t), dataSize * dimension, f))  throw std::runtime_error("value cannot be read");
	return data;
}

template<typename T>
T lexical_cast(const std::string& x)
{
	std::istringstream stream(x);
	T res;
	stream >> res;
	return res;
}

void save_results(const std::string& means_file_name, const std::string& clusters_file_name, const uint32_t meansSize, const value_t* means, const uint32_t dataSize, const value_t* data, const uint32_t* assignedClusters)
{
	FILE* f = fopen(means_file_name.c_str(), "wb");
	if (!f) throw std::runtime_error("cannot open file for writing");
	if (!fwrite(&dimension, sizeof(uint64_t), 1, f)) throw std::runtime_error("dimension cannot be written");
	//if (!fwrite(means, sizeof(value_t), dimension * meansSize, f)) throw std::runtime_error("value cannot be written");
	for (size_t i = 0; i < meansSize; i++)
	{
		if (!fwrite(&means[i*dimension], sizeof(value_t), dimension, f)) throw std::runtime_error("value cannot be written");
		if (!fwrite(&i, sizeof(unsigned char), 1, f)) throw std::runtime_error("value cannot be written");
	}
	if (fclose(f)) throw std::runtime_error("closing the file failed");


	f = fopen(clusters_file_name.c_str(), "wb");
	if (!f) throw std::runtime_error("cannot open file for writing");
	if (!fwrite(&dimension, sizeof(uint64_t), 1, f)) throw std::runtime_error("dimension cannot be written");
	for (size_t i = 0; i < dataSize; i++)
	{
		if (!fwrite(&data[i*dimension], sizeof(value_t), dimension, f)) throw std::runtime_error("value cannot be written");
		if (!fwrite(&assignedClusters[i], sizeof(unsigned char), 1, f)) throw std::runtime_error("value cannot be written");
		//if (!fwrite(&i, sizeof(value_t), 1, f)) throw std::runtime_error("distance cannot be written");
	}
	if (fclose(f)) throw std::runtime_error("closing the file failed");
}

int main(int argc, const char* argv[])
{
	if (argc == 6)
	{
		std::string file_name(argv[1]);
		std::string means_file_name(argv[2]);
		std::string clusters_file_name(argv[3]);
		std::string s_k(argv[4]);
		std::string s_iterations(argv[5]);
		uint32_t k = lexical_cast<uint32_t>(s_k);
		uint32_t iterations = lexical_cast<uint32_t>(s_iterations);
		uint64_t dataSize;

		value_t* data = load(file_name, dataSize);
		value_t* means = (value_t*)calloc(k * dimension, sizeof(value_t));
		uint32_t* assignedClusters = (uint32_t*)calloc(dataSize * dimension, sizeof(uint32_t));
		memcpy(means, data, k * dimension * sizeof(value_t));

		// Add vectors in parallel.
		hipError_t cudaStatus = countKMeans(iterations, dataSize, data, k, means, assignedClusters);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		save_results(means_file_name, clusters_file_name, k, means, dataSize, data, assignedClusters);

		free(data);
		free(means);
		free(assignedClusters);

		return 0;
	}
	usage();
	return 1;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t countKMeans(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize, value_t* means, uint32_t* assignedClusters)
{
    value_t* dev_means = 0;
    value_t* dev_data = 0;
	uint32_t* dev_assignedClusters = 0,* dev_test = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_means, meansSize * dimension * sizeof(value_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_data, dataSize * dimension * sizeof(value_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_assignedClusters, dataSize * sizeof(uint32_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_test, meansSize * sizeof(uint32_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_means, means, meansSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_data, data, dataSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	uint32_t* test = (uint32_t*)calloc(meansSize, sizeof(uint32_t));

    // Launch a kernel on the GPU with one thread for each element.
	int blockSizeN = 32;
	int nBlocksN = (dataSize - 1) / blockSizeN + 1;
	int nBlocksM = (meansSize - 1) / 16 + 1;
	for (uint32_t i = 0; i < iterations; ++i)
	{
		findNearestClusterKernel << <nBlocksN, blockSizeN >> >(meansSize, dev_means, dataSize, dev_data, dev_assignedClusters, dimension);
		hipDeviceSynchronize();
		countNewMeansKernel << <16, nBlocksM >> >(dev_assignedClusters, dataSize, dev_data, dev_means, dimension, dev_test);
		hipDeviceSynchronize();
		cudaStatus = hipMemcpy(test, dev_test, meansSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
		std::vector<uint32_t> t(test, test + meansSize);
	}

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
	}

	cudaStatus = hipMemcpy(means, dev_means, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(assignedClusters, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

   

Error:
    hipFree(dev_data);
    hipFree(dev_means);
	hipFree(dev_assignedClusters);
    
    return cudaStatus;
}
