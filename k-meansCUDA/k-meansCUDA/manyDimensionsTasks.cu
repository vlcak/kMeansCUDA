#include "hip/hip_runtime.h"
#include "manyDimensionsTasks.cuh"
#include "manyDimensionsKernels.cuh"
#include "atomicKernels.cuh"

#include <time.h>
#include <stdio.h>
#include <iostream>

hipError_t countKMeansManyDims(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize, value_t* means, uint32_t* assignedClusters, uint64_t dimension)
{
	value_t* dev_means = 0, *dev_data = 0, *dev_meansSums = 0, *dev_temp = 0;
	uint32_t* dev_assignedClusters = 0, *dev_counts = 0;
	hipError_t cudaStatus;

	const int blockSizeN = BLOCK_SIZE;
	const int pointsPerBlock = BLOCK_SIZE / WARP_SIZE;
	const int nBlocksN = (dataSize - 1) / pointsPerBlock + 1;
	dim3 blockGrid(WARP_SIZE, pointsPerBlock);

	// for DivMeansKernel
	int meansPerBlock = BLOCK_SIZE / dimension;
	int meansBlocks = (meansSize - 1) / meansPerBlock + 1;

	//uint32_t* testAssigned,* testCounts;
	//value_t* testDistances;

	//testAssigned = (uint32_t*)malloc(dataSize * sizeof(uint32_t));
	//testCounts = (uint32_t*)malloc(meansSize * sizeof(uint32_t));
	//testDistances = (value_t*)malloc(meansSize * dimension * sizeof(value_t));

	clock_t start, end;
	start = clock();

	//std::vector<uint32_t> testVector(meansSize);
	try
	{
		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			throw 1;
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_means, meansSize * dimension * sizeof(value_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}

		cudaStatus = hipMalloc((void**)&dev_meansSums, meansSize * dimension * sizeof(value_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}
		else
		{
			hipMemset(dev_meansSums, 0, meansSize * dimension * sizeof(value_t));
		}

		cudaStatus = hipMalloc((void**)&dev_data, dataSize * dimension * sizeof(value_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}

		cudaStatus = hipMalloc((void**)&dev_assignedClusters, dataSize * sizeof(uint32_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}

		cudaStatus = hipMalloc((void**)&dev_counts, meansSize * sizeof(uint32_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}
		else
		{
			hipMemset(dev_counts, 0, meansSize * sizeof(uint32_t));
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_means, means, meansSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}

		cudaStatus = hipMemcpy(dev_data, data, dataSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}

		//uint32_t* test = (uint32_t*)calloc(meansSize, sizeof(uint32_t));
		//value_t* testMeans = (value_t*)calloc(meansSize * dimension , sizeof(value_t));

		// Launch a kernel on the GPU with one thread for each element.
		//int nBlocksN = (dataSize - 1) / blockSizeN + 1;
		//int blockSizeM = 16;
		//int nBlocksM = (meansSize - 1) / blockSizeM + 1;
		for (uint32_t i = 0; i < iterations; ++i)
		{
			findNearestClusterManyDimUnrolledKernel << <nBlocksN, blockGrid, sizeof(value_t)* blockSizeN >> >(meansSize, dev_means, dev_meansSums, dataSize, dev_data, dev_counts, dev_assignedClusters, dimension);
			//findNearestClusterManyDimShuffleKernel << <nBlocksN, blockGrid>> >(meansSize, dev_means, dev_meansSums, dataSize, dev_data, dev_counts, dev_assignedClusters, dimension);
			hipDeviceSynchronize();
			//hipMemcpy(testAssigned, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
			//std::vector<uint32_t> t(testAssigned, testAssigned + dataSize);
			//hipMemcpy(testDistances, dev_meansSums, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
			//std::vector<value_t> t2(testDistances, testDistances + meansSize);
			//hipMemcpy(testCounts, dev_counts, meansSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
			//std::vector<uint32_t> t3(testCounts, testCounts+ meansSize);

			countDivMeansKernel << <meansBlocks, meansPerBlock * dimension >> >(meansSize, dev_counts, dev_means, dev_meansSums, dimension, meansPerBlock);
			hipDeviceSynchronize();

			hipMemset(dev_meansSums, 0, meansSize * dimension * sizeof(value_t));
			hipMemset(dev_counts, 0, meansSize * sizeof(uint32_t));
		}

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			throw 1;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			throw 1;
		}

		cudaStatus = hipMemcpy(means, dev_means, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}

		cudaStatus = hipMemcpy(assignedClusters, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}
	}
	catch (...)
	{
		hipFree(dev_data);
		hipFree(dev_means);
		hipFree(dev_meansSums);
		hipFree(dev_assignedClusters);
		hipFree(dev_counts);
	}

	end = clock();
	std::cout << "Time required for execution: "
		<< (double)(end - start) / CLOCKS_PER_SEC
		<< " seconds." << "\n\n";

	return cudaStatus;
}