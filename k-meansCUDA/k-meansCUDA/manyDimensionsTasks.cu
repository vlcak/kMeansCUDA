#include "hip/hip_runtime.h"
#include "manyDimensionsTasks.cuh"
#include "manyDimensionsKernels.cuh"
#include "atomicKernels.cuh"
#include "helpers.h"

#include <time.h>
#include <stdio.h>
#include <iostream>

hipError_t countKMeansManyDims(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize_u32, value_t* means, uint32_t* assignedClusters, uint64_t dimension_u64, std::string version)
{
    value_t* dev_means = 0, *dev_data = 0, *dev_meansSums = 0;//, *dev_temp = 0;
    uint32_t* dev_assignedClusters = 0, *dev_counts = 0;
    const my_size_t dimension = static_cast<my_size_t>(dimension_u64);
    const my_size_t meansSize = static_cast<my_size_t>(meansSize_u32);
    hipError_t cudaStatus = hipSuccess;

    const int blockSizeN = BLOCK_SIZE;
    const int pointsPerBlock = BLOCK_SIZE / WARP_SIZE;
    const int nBlocksN = (dataSize - 1) / pointsPerBlock + 1;
    dim3 blockGrid(WARP_SIZE, pointsPerBlock);

    // for DivMeansKernel
	int meansPerBlock = BLOCK_SIZE > dimension ? BLOCK_SIZE / dimension : 1;
    int meansBlocks = (meansSize - 1) / meansPerBlock + 1;

    //uint32_t* testAssigned,* testCounts;
    //value_t* testDistances;

    //testAssigned = (uint32_t*)malloc(dataSize * sizeof(uint32_t));
    //testCounts = (uint32_t*)malloc(meansSize * sizeof(uint32_t));
    //testDistances = (value_t*)malloc(meansSize * dimension * sizeof(value_t));

    clock_t start, end;
    start = clock();

    //std::vector<uint32_t> testVector(meansSize);
    try
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        setDevice(DEVICE_ID);

        // Allocate GPU buffers for three vectors (two input, one output)    .
        allocateMemory((void**)&dev_means, meansSize * dimension * sizeof(value_t));

        allocateAndSetMemory((void**)&dev_meansSums, meansSize * dimension * sizeof(value_t), 0);

        allocateMemory((void**)&dev_data, dataSize * dimension * sizeof(value_t));

        allocateMemory((void**)&dev_assignedClusters, dataSize * sizeof(uint32_t));

        allocateAndSetMemory((void**)&dev_counts, meansSize * sizeof(uint32_t), 0);

        // Copy input vectors from host memory to GPU buffers.
        copyMemory(dev_means, means, meansSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);

        copyMemory(dev_data, data, dataSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);

        //uint32_t* test = (uint32_t*)calloc(meansSize, sizeof(uint32_t));
        //value_t* testMeans = (value_t*)calloc(meansSize * dimension , sizeof(value_t));

        // Launch a kernel on the GPU with one thread for each element.
        //int nBlocksN = (dataSize - 1) / blockSizeN + 1;
        //int blockSizeM = 16;
        //int nBlocksM = (meansSize - 1) / blockSizeM + 1;

        auto findNearestClusterKernel = &findNearestClusterManyDimKernel;
        int sharedMemorySize = sizeof(value_t) * blockSizeN;
        if (version == "--unrolled")
        {
            findNearestClusterKernel = &findNearestClusterManyDimUnrolledKernel;
            sharedMemorySize = sizeof(value_t) * blockSizeN;
            std::cout << "Unrolled kernel" << std::endl;
        }
#if __CUDA_ARCH__ >= 300
        if (version == "--shuffle")
        {
            findNearestClusterKernel = &findNearestClusterManyDimShuffleKernel;
            sharedMemorySize = 0;
            std::cout << "Shuffle kernel" << std::endl;
        }
#endif

        std::cout << "Starting execution" << std::endl;
        for (uint32_t i = 0; i < iterations; ++i)
        {
            findNearestClusterKernel << <nBlocksN, blockGrid, sharedMemorySize >> >(meansSize, dev_means, dev_meansSums, dev_data, dev_counts, dev_assignedClusters, dimension);
            synchronizeDevice();
            //hipMemcpy(testAssigned, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
            //std::vector<uint32_t> t(testAssigned, testAssigned + dataSize);
            //hipMemcpy(testDistances, dev_meansSums, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
            //std::vector<value_t> t2(testDistances, testDistances + meansSize);
            //hipMemcpy(testCounts, dev_counts, meansSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
            //std::vector<uint32_t> t3(testCounts, testCounts+ meansSize);

            countDivMeansKernel << <meansBlocks, meansPerBlock * dimension >> >(dev_counts, dev_means, dev_meansSums, dimension, meansPerBlock);
            synchronizeDevice();

            hipMemset(dev_meansSums, 0, meansSize * dimension * sizeof(value_t));
            hipMemset(dev_counts, 0, meansSize * sizeof(uint32_t));
        }

        // Check for any errors launching the kernel
        checkErrors();

        copyMemory(means, dev_means, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);

        copyMemory(assignedClusters, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
    }
    catch (ICUDAException &e)
    {
        fprintf(stderr, "CUDA exception: %s\n", e.what());
        cudaStatus = e.getError();
    }
    catch (std::exception &e)
    {
        fprintf(stderr, "STD exception: %s\n", e.what());
        cudaStatus = hipGetLastError();
    }

    hipFree(dev_data);
    hipFree(dev_means);
    hipFree(dev_meansSums);
    hipFree(dev_assignedClusters);
    hipFree(dev_counts);

    end = clock();
    std::cout << "Time required for execution: "
        << (double)(end - start) / CLOCKS_PER_SEC
        << " seconds." << "\n\n";

    return cudaStatus;
}