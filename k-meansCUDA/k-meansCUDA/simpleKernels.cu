#include "hip/hip_runtime.h"
#include "simpleKernels.cuh"

__global__ void findNearestClusterKernel(const my_size_t meansSize, const value_t *means, const value_t* data, uint32_t* assignedClusters, const my_size_t dimension)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	value_t minDistance = LLONG_MAX, distance = 0, difference = 0;
	for (my_size_t i = 0; i < meansSize; ++i)
	{
		distance = 0;
		for (my_size_t j = 0; j < dimension; ++j)
		{
			difference = means[i * dimension + j] - data[id * dimension + j];
			distance += difference * difference;
		}
		if (minDistance > distance)
		{
			minDistance = distance;
			assignedClusters[id] = i;
		}
	}
}

__global__ void countNewMeansKernel(uint32_t* assignedClusters, const my_size_t dataSize, const value_t* data, value_t* means, const my_size_t dimension, uint32_t* test)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int idOffset = id * dimension;
	uint32_t count = 0;
	for (my_size_t i = idOffset; i < idOffset + dimension; ++i)
	{
		means[i] = 0;
	}
	for (my_size_t i = 0; i < dataSize; ++i)
	{
		if (assignedClusters[i] == id)
		{
			for (my_size_t j = 0; j < dimension; ++j)
			{
				means[idOffset + j] += data[i * dimension + j];
			}
			++count;
		}
	}
	for (my_size_t i = idOffset; i < idOffset + dimension; ++i)
	{
		means[i] /= count;
	}
	test[id] = count;
}