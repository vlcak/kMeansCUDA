#include "hip/hip_runtime.h"
#include "simpleTasks.cuh"
#include "simpleKernels.cuh"

#include <time.h>
#include <stdio.h>
#include <iostream>

hipError_t countKMeansSimple(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize, value_t* means, uint32_t* assignedClusters, uint64_t dimension)
{
	value_t* dev_means = 0;
	value_t* dev_data = 0;
	uint32_t* dev_assignedClusters = 0, *dev_test = 0;
	hipError_t cudaStatus;

	// Launch a kernel on the GPU with one thread for each element.
	int blockSizeN = BLOCK_SIZE;
	int nBlocksN = (dataSize - 1) / blockSizeN + 1;
	int blockSizeM = 16;
	int nBlocksM = (meansSize - 1) / blockSizeM + 1;

	clock_t start, end;
	start = clock();

	try
	{
		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			throw 1;
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_means, meansSize * dimension * sizeof(value_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}

		cudaStatus = hipMalloc((void**)&dev_data, dataSize * dimension * sizeof(value_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}

		cudaStatus = hipMalloc((void**)&dev_assignedClusters, dataSize * sizeof(uint32_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}

		cudaStatus = hipMalloc((void**)&dev_test, meansSize * sizeof(uint32_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_means, means, meansSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}

		cudaStatus = hipMemcpy(dev_data, data, dataSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}

		//uint32_t* test = (uint32_t*)calloc(meansSize, sizeof(uint32_t));

		for (uint32_t i = 0; i < iterations; ++i)
		{
			findNearestClusterKernel << <nBlocksN, blockSizeN >> >(meansSize, dev_means, dataSize, dev_data, dev_assignedClusters, dimension);
			hipDeviceSynchronize();
			countNewMeansKernel << <nBlocksM, blockSizeM >> >(dev_assignedClusters, dataSize, dev_data, dev_means, dimension, dev_test);
			hipDeviceSynchronize();
			//cudaStatus = hipMemcpy(test, dev_test, meansSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
			//std::vector<uint32_t> t(test, test + meansSize);
		}

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			throw 1;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			throw 1;
		}

		cudaStatus = hipMemcpy(means, dev_means, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}

		cudaStatus = hipMemcpy(assignedClusters, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}



	}
	catch (...)
	{
		hipFree(dev_data);
		hipFree(dev_means);
		hipFree(dev_assignedClusters);
	}

	end = clock();
	std::cout << "Time required for execution: "
		<< (double)(end - start) / CLOCKS_PER_SEC
		<< " seconds." << "\n\n";

	return cudaStatus;
}