#include "hip/hip_runtime.h"
#include "warpPerMeanTasks.cuh"
#include "warpPerMeanKernel.cuh"
#include "atomicKernels.cuh"
#include "helpers.h"

#include <time.h>
#include <stdio.h>
#include <iostream>


hipError_t countKMeansWarpPerMean(const uint32_t iterations, const uint32_t dataSize_u32, const value_t* data, const uint32_t meansSize_u32, value_t* means, uint32_t* assignedClusters, uint64_t dimension_u64, std::string version)
{
    value_t *dev_means = 0, *dev_data = 0, *dev_distances = 0;// , *dev_temp = 0;
    uint32_t *dev_assignedClusters = 0, *dev_counts = 0, *dev_locks = 0;
    const my_size_t dataSize = static_cast<my_size_t>(dataSize_u32);
    const my_size_t meansSize = static_cast<my_size_t>(meansSize_u32);
    const my_size_t dimension = static_cast<my_size_t>(dimension_u64);
    hipError_t cudaStatus = hipSuccess;

    // Launch a kernel on the GPU with one thread for each element.
    dim3 blockSizeN(BLOCK_SIZE, 1);
    int nBlocksN = meansSize;
    auto findNearestClusterKernel = &findNearestClusterWarpPerMeanThreadPerPointKernel;
#if __CUDA_ARCH__ >= 300
    if (version == "--dimension")
    {
        blockSizeN = dim3(dimension, BLOCK_SIZE / dimension);
        nBlocksN = (meansSize - 1) / blockSizeN.y + 1;
        findNearestClusterKernel = &findNearestClusterWarpPerMeanThreadPerDimensionKernel;
        std::cout << "Thread per dimension" << std::endl;
    }
#endif

    // for DivMeansKernel
    dim3 blockSizeMeans(dimension, BLOCK_SIZE / dimension);

    int gridSizeMeans = (meansSize - 1) / blockSizeMeans.y + 1;

    clock_t start, end;
    start = clock();


    //std::vector<uint32_t> testVector(meansSize);

    try
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        setDevice(DEVICE_ID);

        // Allocate GPU buffers for three vectors (two input, one output)    .
        allocateMemory((void**)&dev_means, meansSize * dimension * sizeof(value_t));

        allocateAndSetMemory((void**)&dev_distances, dataSize * sizeof(value_t), INT32_MAX);

        allocateAndSetMemory((void**)&dev_locks, dataSize * sizeof(uint32_t), INT32_MAX);

        allocateMemory((void**)&dev_data, dataSize * dimension * sizeof(value_t));

        allocateMemory((void**)&dev_assignedClusters, dataSize * sizeof(uint32_t));

        allocateAndSetMemory((void**)&dev_counts, meansSize * sizeof(uint32_t), 0);

        // Copy input vectors from host memory to GPU buffers.
        copyMemory(dev_means, means, meansSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);

        copyMemory(dev_data, data, dataSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);

        //uint32_t* test = (uint32_t*)calloc(meansSize, sizeof(uint32_t));
        //value_t* testMeans = (value_t*)calloc(meansSize * dimension , sizeof(value_t));

        //int blockSizeM = 16;
        //int nBlocksM = (meansSize - 1) / blockSizeM + 1;

        std::cout << "Starting execution" << std::endl;
        for (uint32_t i = 0; i < iterations; ++i)
        {
            findNearestClusterKernel << <nBlocksN, blockSizeN >> >(dev_means, dataSize, dev_data, dev_locks, dev_distances, dev_assignedClusters, dimension);
            synchronizeDevice();
            countNewMeansWarpPerMeansKernel << <gridSizeMeans, blockSizeMeans >> >(dev_means, dataSize, dev_data, dev_assignedClusters, dimension);
            synchronizeDevice();

            hipMemset(dev_distances, INT32_MAX, dataSize * sizeof(value_t));
        }

        // Check for any errors launching the kernel
        checkErrors();

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        //if (hipDeviceSynchronize() != hipSuccess) {
        //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        //    throw CUDASyncException(cudaStatus);
        //}

        copyMemory(means, dev_means, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
        copyMemory(assignedClusters, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
    }
    catch (ICUDAException &e)
    {
        fprintf(stderr, "CUDA exception: %s\n", e.what());
        cudaStatus = e.getError();
    }
    catch (std::exception &e)
    {
        fprintf(stderr, "STD exception: %s\n", e.what());
        cudaStatus = hipGetLastError();
    }

    // free memory
    hipFree(dev_means);
    hipFree(dev_distances);
    hipFree(dev_locks);
    hipFree(dev_data);
    hipFree(dev_assignedClusters);
    hipFree(dev_counts);

    end = clock();
    std::cout << "Time required for execution: "
        << (double)(end - start) / CLOCKS_PER_SEC
        << " seconds." << "\n\n";

    return cudaStatus;
}