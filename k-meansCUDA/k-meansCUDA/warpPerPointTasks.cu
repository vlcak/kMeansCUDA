#include "hip/hip_runtime.h"
#include "warpPerPointTasks.cuh"
#include "warpPerPointKernel.cuh"
#include "atomicKernels.cuh"

#include <time.h>
#include <stdio.h>
#include <iostream>


hipError_t countKMeansWarpPerPoint(const uint32_t iterations, const uint32_t dataSize, const value_t* data, const uint32_t meansSize, value_t* means, uint32_t* assignedClusters, uint64_t dimension)
{
	value_t* dev_means = 0, *dev_data = 0, *dev_meansSums = 0, *dev_temp = 0;
	uint32_t* dev_assignedClusters = 0, *dev_counts = 0;
	hipError_t cudaStatus;

	// Launch a kernel on the GPU with one thread for each element.
	int pointsPerWarp = BLOCK_SIZE / meansSize;
	dim3 blockSizeN = (meansSize, pointsPerWarp);
	int nBlocksN = (dataSize - 1) / pointsPerWarp + 1;

	// for DivMeansKernel
	int meansPerBlock = BLOCK_SIZE / dimension;
	int meansBlocks = (meansSize - 1) / meansPerBlock + 1;

	int sharedMemomrySize = sizeof(value_t)* (dimension * pointsPerWarp + blockSizeN.x * blockSizeN.y);

	clock_t start, end;
	start = clock();

	//std::vector<uint32_t> testVector(meansSize);

	try
	{
		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			throw 1;
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_means, meansSize * dimension * sizeof(value_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}

		cudaStatus = hipMalloc((void**)&dev_meansSums, meansSize * dimension * sizeof(value_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}
		else
		{
			hipMemset(dev_meansSums, 0, meansSize * dimension * sizeof(value_t));
		}

		cudaStatus = hipMalloc((void**)&dev_data, dataSize * dimension * sizeof(value_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}

		cudaStatus = hipMalloc((void**)&dev_assignedClusters, dataSize * sizeof(uint32_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}

		cudaStatus = hipMalloc((void**)&dev_counts, meansSize * sizeof(uint32_t));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw 1;
		}
		else
		{
			hipMemset(dev_counts, 0, meansSize * sizeof(uint32_t));
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_means, means, meansSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}

		cudaStatus = hipMemcpy(dev_data, data, dataSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}

		//uint32_t* test = (uint32_t*)calloc(meansSize, sizeof(uint32_t));
		//value_t* testMeans = (value_t*)calloc(meansSize * dimension , sizeof(value_t));

		//int blockSizeM = 16;
		//int nBlocksM = (meansSize - 1) / blockSizeM + 1;
		for (uint32_t i = 0; i < iterations; ++i)
		{
			findNearestWarpPerPointKernel << <nBlocksN, blockSizeN, sharedMemomrySize >> >(meansSize, dev_means, dev_meansSums, dataSize, dev_data, dev_counts, dimension, 0, dataSize);
			hipDeviceSynchronize();
			countDivMeansKernel << <meansBlocks, meansPerBlock * dimension >> >(meansSize, dev_counts, dev_means, dev_meansSums, dimension, meansPerBlock);
			hipDeviceSynchronize();

			hipMemset(dev_meansSums, 0, meansSize * dimension * sizeof(value_t));
			hipMemset(dev_counts, 0, meansSize * sizeof(uint32_t));
		}

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			throw 1;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			throw 1;
		}

		cudaStatus = hipMemcpy(means, dev_means, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}

		cudaStatus = hipMemcpy(assignedClusters, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw 1;
		}
	}
	catch (...)
	{
		hipFree(dev_data);
		hipFree(dev_means);
		hipFree(dev_meansSums);
		hipFree(dev_assignedClusters);
		hipFree(dev_counts);
	}

	end = clock();
	std::cout << "Time required for execution: "
		<< (double)(end - start) / CLOCKS_PER_SEC
		<< " seconds." << "\n\n";

	return cudaStatus;
}