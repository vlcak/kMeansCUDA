#include "hip/hip_runtime.h"
#include "warpPerPointTasks.cuh"
#include "warpPerPointKernel.cuh"
#include "atomicKernels.cuh"
#include "helpers.h"

#include <time.h>
#include <stdio.h>
#include <iostream>


hipError_t countKMeansWarpPerPoint(const uint32_t iterations, const uint32_t dataSize_u32, const value_t* data, const uint32_t meansSize_u32, value_t* means, uint32_t* assignedClusters, uint64_t dimension_u64, std::string version)
{
    value_t* dev_means = 0, *dev_data = 0, *dev_meansSums = 0;//, *dev_temp = 0;
    uint32_t* dev_assignedClusters = 0, *dev_counts = 0;
    const my_size_t dataSize = static_cast<my_size_t>(dataSize_u32);
    const my_size_t meansSize = static_cast<my_size_t>(meansSize_u32);
    const my_size_t dimension = static_cast<my_size_t>(dimension_u64);
    hipError_t cudaStatus = hipSuccess;

    // Launch a kernel on the GPU with one thread for each element.
	int pointsPerBlock= BLOCK_SIZE > meansSize ? WARP_SIZE / meansSize : 1;
	dim3 blockSizeN(meansSize, pointsPerBlock);
	int nBlocksN = (dataSize - 1) / pointsPerBlock + 1;
    auto findNearestClusterKernel = &findNearestWarpPerPointKernel;
    int sharedMemomrySize = sizeof(value_t)* (/*dimension * pointsPerWarp + */blockSizeN.x * blockSizeN.y);
    if (version == "--sharedMemory")
    {
        findNearestClusterKernel = &findNearestWarpPerPointSMKernel;
		sharedMemomrySize = sizeof(value_t)* (dimension * pointsPerBlock + blockSizeN.x * blockSizeN.y);
        std::cout << "Shared memory" << std::endl;
    }
#if __CUDA_ARCH__ >= 300
    if (version == "--shuffle")
    {
        findNearestClusterKernel = &findNearestWarpPerPointShuffleKernel;
        sharedMemomrySize = 0;
        std::cout << "Shuffle" << std::endl;
    }
#endif

    // for DivMeansKernel
	int meansPerBlock = BLOCK_SIZE > dimension ? BLOCK_SIZE / dimension : 1;
    int meansBlocks = (meansSize - 1) / meansPerBlock + 1;


    clock_t start, end;
    start = clock();

    //std::vector<uint32_t> testVector(meansSize);

    try
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        setDevice(DEVICE_ID);

        // Allocate GPU buffers for three vectors (two input, one output)    .
        allocateMemory((void**)&dev_means, meansSize * dimension * sizeof(value_t));

        allocateAndSetMemory((void**)&dev_meansSums, meansSize * dimension * sizeof(value_t), 0);

        allocateMemory((void**)&dev_data, dataSize * dimension * sizeof(value_t));

        allocateMemory((void**)&dev_assignedClusters, dataSize * sizeof(uint32_t));

        allocateAndSetMemory((void**)&dev_counts, meansSize * sizeof(uint32_t), 0);

        // Copy input vectors from host memory to GPU buffers.
        copyMemory(dev_means, means, meansSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);
        copyMemory(dev_data, data, dataSize * dimension * sizeof(value_t), hipMemcpyHostToDevice);

        //uint32_t* test = (uint32_t*)calloc(meansSize, sizeof(uint32_t));
        //value_t* testMeans = (value_t*)calloc(meansSize * dimension , sizeof(value_t));

        //int blockSizeM = 16;
        //int nBlocksM = (meansSize - 1) / blockSizeM + 1;

        std::cout << "Starting execution" << std::endl;
        for (int32_t i = 0; i < iterations; ++i)
        {
            findNearestClusterKernel << <nBlocksN, blockSizeN, sharedMemomrySize >> >(dev_means, dev_meansSums, dev_data, dev_counts, dimension);
            synchronizeDevice();
            countDivMeansKernel << <meansBlocks, meansPerBlock * dimension >> >(dev_counts, dev_means, dev_meansSums, dimension, meansPerBlock);
            synchronizeDevice();

            hipMemset(dev_meansSums, 0, meansSize * dimension * sizeof(value_t));
            hipMemset(dev_counts, 0, meansSize * sizeof(uint32_t));
        }

        // Check for any errors launching the kernel
        checkErrors();

        copyMemory(means, dev_means, meansSize * dimension * sizeof(value_t), hipMemcpyDeviceToHost);
        copyMemory(assignedClusters, dev_assignedClusters, dataSize * sizeof(uint32_t), hipMemcpyDeviceToHost);
    }
    catch (ICUDAException &e)
    {
        fprintf(stderr, "CUDA exception: %s\n", e.what());
        cudaStatus = e.getError();
    }
    catch (std::exception &e)
    {
        fprintf(stderr, "STD exception: %s\n", e.what());
        cudaStatus = hipGetLastError();
    }

    hipFree(dev_data);
    hipFree(dev_means);
    hipFree(dev_meansSums);
    hipFree(dev_assignedClusters);
    hipFree(dev_counts);

    end = clock();
    std::cout << "Time required for execution: "
        << (double)(end - start) / CLOCKS_PER_SEC
        << " seconds." << "\n\n";

    return cudaStatus;
}